#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "featuredistribute_cuda_kernel.h"

__global__ void featuredistribute_cuda_kernel(int b, int n, int m, const float *max_xyz, const float *xyz, int *distribute_idx) {
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    max_xyz += bs_idx * n * 3;
    xyz += bs_idx * m * 3 + pt_idx * 3;
    distribute_idx += bs_idx * m + pt_idx;

    float x = xyz[0];
    float y = xyz[1];
    float z = xyz[2];

    float min_dist2 = 100000;
    int min_dist_idx = -1;
    for (int k = 0; k < n; ++k) {
        float max_x = max_xyz[k * 3 + 0];
        float max_y = max_xyz[k * 3 + 1];
        float max_z = max_xyz[k * 3 + 2];
        float d2 = (max_x - x) * (max_x - x) + (max_y - y) * (max_y - y) + (max_z - z) * (max_z - z);
        if (d2 < min_dist2){
            min_dist_idx = k;
            min_dist2 = d2;
        }
    }
    distribute_idx[0] = min_dist_idx;
}


void featuredistribute_cuda_launcher(int b, int n, int m, const float *max_xyz, const float *xyz, int *distribute_idx, hipStream_t stream) {
    // param max_xyz: (b, n, 3)
    // param xyz: (b, m, 3)
    // return distribute_idx: (b, m)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    featuredistribute_cuda_kernel<<<blocks, threads, 0, stream>>>(b, n, m, max_xyz, xyz, distribute_idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

__global__ void featuregather_forward_cuda_kernel(int b, int n, int m, int c, const float *max_feature, const int *distribute_idx, float *distribute_feature) {
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

    max_feature += bs_idx * c * n + c_idx * n;
    distribute_idx += bs_idx * m + pt_idx;
    distribute_feature += bs_idx * c * m + c_idx * m + pt_idx;

    int idx = distribute_idx[0];
    distribute_feature[0] = max_feature[idx];
}


void featuregather_forward_cuda_launcher(int b, int n, int m, int c, const float *max_feature, const int *distribute_idx, float *distribute_feature, hipStream_t stream){
    // param max_feature: (b, c, n)
    // param distribute_idx: (b, m)
    // return distribute_feature: (b, c, m)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    featuregather_forward_cuda_kernel<<<blocks, threads, 0, stream>>>(b, n, m, c, max_feature, distribute_idx, distribute_feature);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void featuregather_backward_cuda_kernel(int b, int n, int m, int c, const float *grad_distribute_feature, const int *distribute_idx, float *grad_max_feature){
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(bs_idx >= b || c_idx >= c || pt_idx >= m) return;

    grad_distribute_feature += bs_idx * c * m + c_idx * m + pt_idx;
    distribute_idx += bs_idx * m + pt_idx;
    grad_max_feature += bs_idx * c * n + c_idx * n;

    int idx = distribute_idx[0];
    atomicAdd(grad_max_feature + idx, grad_distribute_feature[0]);
}


void featuregather_backward_cuda_launcher(int b, int n, int m, int c, const float *grad_distribute_feature, const int *distribute_idx, float *grad_max_feature, hipStream_t stream){
    // param grad_distribute_feature: (b, c, m)
    // param distribute_idx: (b, m)
    // return grad_max_feature: (b, c, n)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    featuregather_backward_cuda_kernel<<<blocks, threads, 0, stream>>>(b, n, m, c, grad_distribute_feature, distribute_idx, grad_max_feature);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}