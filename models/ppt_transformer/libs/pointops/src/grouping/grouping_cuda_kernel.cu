#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "grouping_cuda_kernel.h"

// input: points(b, c, n) idx(b, m, nsample)
// output: out(b, c, m, nsample)
__global__ void grouping_forward_cuda_kernel(int b, int c, int n, int m, int nsample, const float *points, const int *idx, float *out)
{
    int batch_index = blockIdx.x;
    points += batch_index * n * c;
    idx += batch_index * m * nsample;
    out += batch_index * m * nsample * c;
    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * m; i += stride)
    {
        const int l = i / m;
        const int j = i % m;
        for (int k = 0; k < nsample; ++k)
        {
            int ii = idx[j * nsample + k];
            out[(l * m + j) * nsample + k] = points[l * n + ii];
        }
    }
}

// input: grad_out(b, c, m, nsample), idx(b, m, nsample)
// output: grad_points(b, c, n)
__global__ void grouping_backward_cuda_kernel(int b, int c, int n, int m, int nsample, const float *grad_out, const int *idx, float *grad_points)
{
    int batch_index = blockIdx.x;
    grad_out += batch_index * m * nsample * c;
    idx += batch_index * m * nsample;
    grad_points += batch_index * n * c;
    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * m; i += stride)
    {
        const int l = i / m;
        const int j = i % m;
        for (int k = 0; k < nsample; ++k)
        {
            int ii = idx[j * nsample + k];
            atomicAdd(grad_points + l * n + ii, grad_out[(l * m + j) * nsample + k]);
        }
    }
}

void grouping_forward_cuda_launcher(int b, int c, int n, int m, int nsample, const float *points, const int *idx, float *out)
{
    grouping_forward_cuda_kernel<<<b, opt_block_config(m, c), 0>>>(b, c, n, m, nsample, points, idx, out);
}

void grouping_backward_cuda_launcher(int b, int c, int n, int m, int nsample, const float *grad_out, const int *idx, float *grad_points)
{
    grouping_backward_cuda_kernel<<<b, opt_block_config(m, c), 0>>>(b, c, n, m, nsample, grad_out, idx, grad_points);
}

// input: points(b, c, n) idx(b, npoints, nsample)
// output: out(b, c, npoints, nsample)
__global__ void grouping_forward_cuda_kernel_fast(int b, int c, int n, int npoints, int nsample, const float *__restrict__ points, const int *__restrict__ idx, float *__restrict__ out) {
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;

    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;
    int in_idx = bs_idx * c * n + c_idx * n + idx[0];
    int out_idx = bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx;

    out[out_idx] = points[in_idx];
}

// input: points(b, c, n) idx(b, npoints, nsample)
// output: out(b, c, npoints, nsample)
void grouping_forward_cuda_launcher_fast(int b, int c, int n, int npoints, int nsample, const float *points, const int *idx, float *out) {

    hipError_t err;

    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    grouping_forward_cuda_kernel_fast<<<blocks, threads, 0>>>(b, c, n, npoints, nsample, points, idx, out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


