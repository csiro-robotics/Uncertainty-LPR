#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "ballquery_cuda_kernel.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
__global__ void ballquery_cuda_kernel(int b, int n, int m, float radius, int nsample, const float *new_xyz, const float *xyz, int *idx)
{
    int batch_index = blockIdx.x;
    xyz += batch_index * n * 3;
    new_xyz += batch_index * m * 3;
    idx += m * nsample * batch_index;
    int index = threadIdx.x;
    int stride = blockDim.x;

    float radius2 = radius * radius;
    for (int j = index; j < m; j += stride)
    {
        float new_x = new_xyz[j * 3 + 0];
        float new_y = new_xyz[j * 3 + 1];
        float new_z = new_xyz[j * 3 + 2];
        for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k)
        {
            float x = xyz[k * 3 + 0];
            float y = xyz[k * 3 + 1];
            float z = xyz[k * 3 + 2];
            float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
            if (d2 < radius2)
            {
                if (cnt == 0)
                {
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                ++cnt;
            }
        }
    }
}

void ballquery_cuda_launcher(int b, int n, int m, float radius, int nsample, const float *new_xyz, const float *xyz, int *idx)
{
    ballquery_cuda_kernel<<<b, opt_n_threads(m), 0>>>(b, n, m, radius, nsample, new_xyz, xyz, idx);
}


__global__ void ballquery_cuda_kernel_fast(int b, int n, int m, float radius, int nsample, const float *__restrict__ new_xyz, const float *__restrict__ xyz, int *__restrict__ idx) {
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    idx += bs_idx * m * nsample + pt_idx * nsample;

    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float z = xyz[k * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
        if (d2 < radius2){
            if (cnt == 0){
                for (int l = 0; l < nsample; ++l) {
                    idx[l] = k;
                }
            }
            idx[cnt] = k;
            ++cnt;
            if (cnt >= nsample){
                break;
            }
        }
    }
}


void ballquery_cuda_launcher_fast(int b, int n, int m, float radius, int nsample, const float *new_xyz, const float *xyz, int *idx, hipStream_t stream) {
    // param new_xyz: (B, m, 3)
    // param xyz: (B, n, 3)
    // param idx: (B, m, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    ballquery_cuda_kernel_fast<<<blocks, threads, 0, stream>>>(b, n, m, radius, nsample, new_xyz, xyz, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
